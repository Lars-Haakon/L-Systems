#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include "cudainfo.cuh"

extern void PrintCudaInfo()
{
    int n_devices;
    hipGetDeviceCount(&n_devices);
    printf("Number of CUDA devices: %d\n", n_devices);
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, 0);
    printf("CUDA device name: %s\n" , device_prop.name);
}

__global__ void Fill(char* device_module, float* transform, int *count)
{
    int tId = threadIdx.x;

    if(device_module[tId] == 'F')
    {
        
    }
}

__global__ void HillisSteeleScan(float* transform, int *count, int n)
{
    extern __shared__ int temp[]; // allocated on invocation

    int tId = threadIdx.x;
    int pout = 0, pin = 1;
    // Load input into shared memory.
    // This is exclusive scan, so shift right by one
    // and set first element to 0
    temp[pout*n + tId] = count[tId]; // inclusive
    //temp[pout*n + tId] = (tId > 0) ? in[tId-1] : 0; // exclusive
    __syncthreads();
    for (int offset = 1; offset < n; offset *= 2)
    {
        pout = 1 - pout; // swap double buffer indices
        pin = 1 - pout;

        temp[pout*n+tId] = temp[pin*n+tId] + ((tId >= offset)?temp[pin*n+tId - offset]:0);

        float* A = &transform[16*(pin*n + tId)];
        float* B = &transform[16*(pin*n + tId - offset)];
        float* C = &transform[16*(pout*n + tId)];

        for (int i = 0; i < 4; i++)
    	{
    		for (int j = 0; j < 4; j++)
    		{
    			C[i*4 + j] =	A[i*4+0] * ((tId >= offset)?B[0*4+j]:0) +
    							A[i*4+1] * ((tId >= offset)?B[1*4+j]:0) +
    							A[i*4+2] * ((tId >= offset)?B[2*4+j]:0) +
    							A[i*4+3] * ((tId >= offset)?B[3*4+j]:0);
    		}
        }

        __syncthreads();
    }

    count[tId] = temp[pout*n+tId]; // write output
    float* A = &transform[16*(pin*n + tId)];
    float* C = &transform[16*tId];
    for(int i = 0; i < 16; i++)
        C[i] = A[i];
}

__global__ void Count(float* device_lookUpTable, char* device_module, float* device_transform, int* device_count)
{
    int bId = blockIdx.x;
    int tId = threadIdx.x;

    device_transform[bId*16 + tId] = device_lookUpTable[device_module[bId]*16 + tId];

    if(tId == 0) // only need 1 thread to set value to avoid conflicts
    {
        device_count[bId] = 0;

        if(device_module[bId] == 'F')
        {
            device_count[bId] = 1;
        }
    }
}

float* device_lookUpTable = 0;
char* device_module = 0;
float* device_transform = 0;
int* device_count = 0;

extern int FillData(float* lookUpTable, int lookUpTableSize, const char* module, int moduleLength)
{
    // move lookUpTable to device
    hipMalloc((void**)&device_lookUpTable, lookUpTableSize);
    hipMemcpy((void*)device_lookUpTable, (void*)lookUpTable, lookUpTableSize, hipMemcpyHostToDevice);

    hipMalloc((void**)&device_module, moduleLength);
    hipMemcpy((void*)device_module, (void*)module, moduleLength, hipMemcpyHostToDevice);

    hipMalloc((void**)&device_transform, 2*moduleLength*16*sizeof(float)); // multiply by 2 to allocate a double buffer in shared memory

    hipMalloc((void**)&device_count, moduleLength * sizeof(int));

    dim3 numThreadsPerBlock(16);
    dim3 numBlocks(moduleLength);
    Count<<<numBlocks, numThreadsPerBlock>>>(device_lookUpTable, device_module, device_transform, device_count);

    dim3 numThreadsPerBlock2(moduleLength);
    dim3 numBlocks2(1);
    HillisSteeleScan<<<numBlocks2, numThreadsPerBlock2, 2*moduleLength*sizeof(int)>>>(device_transform, device_count, moduleLength);

    /*float* host_matrix = (float*)malloc(16*sizeof(float));
    hipMemcpy(host_matrix, device_transform+7*16, 16*sizeof(float), hipMemcpyDeviceToHost);
    for(int i = 0; i < 4; i++)
    {
        for(int j = 0; j < 4; j++)
        {
            printf("%.2f ", host_matrix[i*4+j]);
        }
        printf("\n");
    }*/
    /*int* host_out = (int*) malloc(moduleLength*sizeof(int));
    hipMemcpy(host_out, device_count, moduleLength*sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0; i < moduleLength; i++)
    {
        printf("%d\n", host_out[i]);
    }*/

    // get the last value
    int size = -1;
    hipMemcpy(&size, device_count + moduleLength-1, sizeof(int), hipMemcpyDeviceToHost);

    return size;
}

extern void FillVBO(unsigned int vbo, int moduleLength)
{
    struct hipGraphicsResource* cudaResource;
    hipGraphicsGLRegisterBuffer(&cudaResource, vbo, cudaGraphicsMapFlagsNone);

    hipGraphicsMapResources(1, &cudaResource, 0);
    float* positions;
    size_t num_bytes;
    hipGraphicsResourceGetMappedPointer((void**)&positions, &num_bytes, cudaResource);

    dim3 numThreadsPerBlock(moduleLength);
    dim3 numBlocks(1);
    Fill<<<numBlocks, numThreadsPerBlock>>>(device_module, device_transform, device_count);

    hipFree(device_lookUpTable);
    hipFree(device_module);
    hipFree(device_transform);
    hipFree(device_count);

    hipGraphicsUnmapResources(1, &cudaResource, 0);
    hipGraphicsUnregisterResource(cudaResource);
}
