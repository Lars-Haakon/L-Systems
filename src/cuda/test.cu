#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "test.cuh"

void PrintCudaInfo()
{
    int n_devices;
    hipGetDeviceCount(&n_devices);
    printf("Number of CUDA devices: %d\n", n_devices);
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, 0);
    printf("CUDA device name: %s\n" , device_prop.name);
}
